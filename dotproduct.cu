/*
 * dotproduct.cu
 * includes setup funtion called from "driver" program
 * also includes kernel function 'kernel_dotproduct[2]()'
 * largely inspired in the pdf http://www.cuvilib.com/Reduction.pdf
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 1024

struct timeval  tp1, tp2;

__global__ void kernel_dotproduct(long long *force_d, long long *distance_d, long long *result_d, long long size) {
    extern __shared__ long long sadata[];
    
    int n = blockDim.x;
    int nTotalThreads;
    if (!n){
	nTotalThreads = n;
    }else{
	//(0 == 2^0)
    	int x = 1;
    	while(x < n)
    	{
      	    x <<= 1;
    	}
        nTotalThreads = x;
    }

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    long long i = blockIdx.x*nTotalThreads + threadIdx.x;
    sadata[tid] = 0;
    if(i < size){
    	sadata[tid] = force_d[i]*distance_d[i];
    }
    __syncthreads();
    
    // do reduction in shared mem
    //if(i < size){
    for (unsigned int s=1; s < nTotalThreads; s *= 2) {
        if (tid % (2*s) == 0) {
            sadata[tid] += sadata[tid + s];
        }
        __syncthreads();
    }
    //}  
    // write result for this block to global mem
    if (tid == 0) result_d[blockIdx.x] = sadata[0];
}

template <unsigned int blockSize>
__global__ void kernel_dotproduct2(long long *force_d, long long *distance_d, long long *result_d, long long size)
{
	extern __shared__ long long sdata[];
	int n = blockDim.x;
    	int nTotalThreads;
    	if (!n){
        	nTotalThreads = n;
    	}else{
        	//(0 == 2^0)
        	int x = 1;
        	while(x < n)
        	{
            		x <<= 1;
        	}
        	nTotalThreads = x;
    	}

	unsigned int tid = threadIdx.x;
	long long i = blockIdx.x*(nTotalThreads*2) + threadIdx.x;
	sdata[tid] = 0;
	if((i+nTotalThreads)< size){
		sdata[tid] = force_d[i]*distance_d[i] + force_d[i+nTotalThreads]*distance_d[i+nTotalThreads] ;
	} else {
		if(i < size){
			sdata[tid] = force_d[i]*distance_d[i];
		}
	}
	__syncthreads();
	for (long long s=nTotalThreads/2; s>32 && (tid+s) < size; s>>=1)
	{
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}

	// write result for this block to global mem
	if (tid == 0) result_d[blockIdx.x] = sdata[0];
	
}

// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void cuda_dotproduct (long long *force, long long *distance, long long arraySize, long long *result_array, double *time_result)
{
	// force_d, distance_d and result_d are the GPU counterparts of the arrays that exists in host memory 
	long long *force_d;
	long long *distance_d;
	long long *result_d;

	hipError_t op_result;

	// Reset the device and exit
    	op_result = hipDeviceReset();

    	if (op_result != hipSuccess)
    	{
        	fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(op_result));
        	exit(EXIT_FAILURE);
    	}	
	// allocate space in the device 
	op_result = hipMalloc ((void**) &force_d, sizeof(long long) * arraySize);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMalloc (foce) failed.");
		exit(1);
	}
	op_result = hipMalloc ((void**) &distance_d, sizeof(long long) * arraySize);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMalloc (distance) failed.");
		exit(1);
	}
	op_result = hipMalloc ((void**) &result_d, sizeof(long long)*arraySize);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipMalloc (result) failed.");
                exit(1);
        }
	
	//copy the arrays from host to the device 
	op_result = hipMemcpy (force_d, force, sizeof(long long) * arraySize, hipMemcpyHostToDevice);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (force) failed.");
		exit(1);
	}
	op_result = hipMemcpy (distance_d, distance, sizeof(long long) * arraySize, hipMemcpyHostToDevice);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (distance) failed.");
		exit(1);
	}
        
	op_result = hipMemcpy (result_d, result_array, sizeof(long long) * arraySize, hipMemcpyHostToDevice);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipMemcpy host->dev (result) failed.");
                exit(1);
        }
	
	int threads;
	if(arraySize < 128){
		threads = 64;
	} else if (arraySize < 256 ){
		threads = 128;
	} else if (arraySize < 512){
		threads = 256;
	} else if (arraySize < 1024){
		threads = 512;
	} else {
		threads = BLOCK_SIZE;
	}
	long long block_size = threads;
        long long blocks = ceil(arraySize / ((float) block_size));
	// set execution configuration
        dim3 dimblock (block_size);
        dim3 dimgrid (blocks);
        int smemSize = dimblock.x * sizeof(long long);
        
	// actual computation: Call the kernel
	gettimeofday(&tp1, NULL);
	switch (threads)
	{
		case 64:
                  kernel_dotproduct2<64><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
                  break;
		case 128:
		  kernel_dotproduct2<128><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
		  break;
		case 256:
                  kernel_dotproduct2<256><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
                  break;
		case 512:
                  kernel_dotproduct2<256><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
                  break;
		default:
		 kernel_dotproduct2<BLOCK_SIZE><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize); 
		 break;
	}
	gettimeofday(&tp2, NULL);
    	*time_result = (double) (tp2.tv_usec - tp1.tv_usec) / 1000000 + (double) (tp2.tv_sec - tp1.tv_sec);

	op_result = hipMemcpy (result_array, result_d, sizeof(long long)*arraySize, hipMemcpyDeviceToHost);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipMemcpy host <- dev (result) failed.");
                exit(1);
        }
	
	// release the memory on the GPU 
	op_result = hipFree (force_d);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipFree (force) failed.");
		exit(1);
	}
	op_result = hipFree (distance_d);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipFree (distance) failed.");
		exit(1);
	}
	op_result = hipFree (result_d);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipFree (distance) failed.");
                exit(1);
        }
        
}

